

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>


#define NUM 10000
#define SEED 18
#define CUDA_ERROR_EXIT(str) do{\
				hipError_t err = hipGetLastError();\
				if(err!=hipSuccess){\
					printf("Cuda Error: %s for %s \n",hipGetErrorString(err),str);\
					exit(-1);\
					}\
				}while(0);

#define TDIFF(start, end) ((end.tv_sec - start.tv_sec) * 1000000UL + (end.tv_usec - start.tv_usec))


__global__ void calculate(int *d_arr,unsigned long num,int merge_size){
	int i=blockIdx.x * blockDim.x + threadIdx.x;
	if(i%merge_size!=0 || i>num)
		return;
	if(i+merge_size/2<num)
		d_arr[i]=d_arr[i]^d_arr[i+merge_size/2];
	return;
}



int main(int argc, char **argv)
{
	struct timeval start, end, t_start, t_end;
	if(argc!=3)
	{
		printf("Insufficient number of arguments");
		exit(-1);
	}
	
	unsigned long num=NUM;
	num = atoi(argv[1]);
	unsigned long seed=SEED;
 	seed = atoi(argv[2]);
	
	srand(seed);

	int *h_arr;
	int *d_arr;
	h_arr=(int *)malloc(num*sizeof(num));
	for(int i=0;i<num;i++)
		h_arr[i]=rand();

	/*	
	h_arr[0]=10;
	h_arr[1]=9;
	h_arr[2]=19;
	h_arr[3]=5;
	h_arr[4]=4;*/
	
	gettimeofday(&t_start,NULL);
	hipMalloc(&d_arr,num*sizeof(int));
	CUDA_ERROR_EXIT("cudamalloc");

	hipMemcpy(d_arr,h_arr,num*sizeof(int),hipMemcpyHostToDevice);
	CUDA_ERROR_EXIT("memcpy");
	
	gettimeofday(&start,NULL);
	int merge_size=1;
	while(merge_size<num){
		merge_size*=2;
		calculate<<< (1023+num)/1024,1024  >>> (d_arr,num,merge_size);
	}
	gettimeofday(&end,NULL);
	
	hipMemcpy(h_arr,d_arr,num*sizeof(int),hipMemcpyDeviceToHost);
	CUDA_ERROR_EXIT("memcpy");
	gettimeofday(&t_end,NULL);	

//	printf("Total time = %ld microsecs Processsing =%ld microsecs\n", TDIFF(t_start, t_end), TDIFF(start, end));
	printf("%d\n",h_arr[0]);

	hipFree(d_arr);
	free(h_arr);
	return 0;	
}	
